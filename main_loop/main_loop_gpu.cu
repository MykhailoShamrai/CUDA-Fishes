#include "hip/hip_runtime.h"
#include "main_loop_gpu.cuh"
#include "../objects/fishes.cuh"
#include "../objects/grid.cuh"
#include "../objects/options.cuh"
#include <>

__global__ void CountForFishes(Grid grid, Options* options, Fishes fishes, float* buffer, int n)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n)
		return;

	fishes.CountForAFish(i, &grid, options);
	// Hardcoded parameters for triangles
	fishes.FindTrianglesForAFish(i, buffer, 10, 6);
}