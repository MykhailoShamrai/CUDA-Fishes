#include "grid.cuh"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../include/helpers.cuh"


Grid::Grid(int nFishes, int radiusForFishes, int width, int heith, bool onGpu):
	onGpu(onGpu), n_fishes(nFishes)
{
	// TODO: Firstly count how many cells there is

	if (onGpu)
	{
		
	}
	else
	{

	}
}

Grid::~Grid()
{
	if (onGpu)
	{
		d_CleanMemory();
	}
	else
	{
		h_CleanMemory();
	}
}

void Grid::h_AllocateMemory()
{
	// Allocate array of ints size number of fishes
	this->cell_id = (int*)malloc(sizeof(int) * this->n_fishes);
	// Allocate array of ints size number of fishes
	this->fish_id - (int*)malloc(sizeof(int) * this->n_fishes);
	// Allocate array of ints size number of cells
	this->cells_starts = (int*)malloc(sizeof(int) * this->n_cells);
	// Allocate array if ints size number of cells
	this->cells_ends = (int*)malloc(sizeof(int) * this->n_cells);
}

void Grid::d_AllocateMemory()
{
	checkCudaErrors(hipMalloc((void**)&this->cell_id, sizeof(int) * this->n_fishes));
	checkCudaErrors(hipMalloc((void**)&this->fish_id, sizeof(int) * this->n_fishes));
	checkCudaErrors(hipMalloc((void**)&this->cells_starts, sizeof(int) * this->n_cells));
	checkCudaErrors(hipMalloc((void**)&this->cells_ends, sizeof(int) * this->n_cells));	
}

void Grid::h_CleanMemory()
{
	free(this->cell_id);
	free(this->fish_id);
	free(this->cells_starts);
	free(this->cells_ends);	
}

void Grid::d_CleanMemory()
{
	checkCudaErrors(hipFree(this->cell_id));
	checkCudaErrors(hipFree(this->fish_id));
	checkCudaErrors(hipFree(this->cells_starts));
	checkCudaErrors(hipFree(this->cells_ends));	
}