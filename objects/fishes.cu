#include "hip/hip_runtime.h"
#include "fishes.cuh"
#include <stdlib.h>
#include "../third_party/cuda-samples/hip/hip_vector_types.h"
#include "grid.cuh"
#include "options.cuh"
#include "../include/helpers.cuh"

Fishes::Fishes(int n, bool onGpu): n(n), onGpu(onGpu)
{
	if (onGpu)
	{
		d_AllocateMemoryForFishes();
	}
	else
	{
		h_AllocateMemoryForFishes();
	}
}


void Fishes::h_AllocateMemoryForFishes()
{
	this->x_before_movement = (float*)malloc(sizeof(float) * n);
	this->y_before_movement = (float*)malloc(sizeof(float) * n);
	this->x_vel_before_movement = (float*)malloc(sizeof(float) * n);
	this->y_vel_before_movement = (float*)malloc(sizeof(float) * n);

	this->x_after_movement = (float*)malloc(sizeof(float) * n);
	this->y_after_movement = (float*)malloc(sizeof(float) * n);
	this->x_vel_after_movement = (float*)malloc(sizeof(float) * n);
	this->y_vel_after_movement = (float*)malloc(sizeof(float) * n);

	this->types = (Fishes::FishType*)malloc(sizeof(Fishes::FishType) * n);
} 

void Fishes::d_AllocateMemoryForFishes()
{
	checkCudaErrors(hipMalloc((void**)&this->x_before_movement, sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&this->y_before_movement, sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&this->x_vel_before_movement, sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&this->y_vel_before_movement, sizeof(float) * n));

	checkCudaErrors(hipMalloc((void**)&this->x_after_movement, sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&this->y_after_movement, sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&this->x_vel_after_movement, sizeof(float) * n));
	checkCudaErrors(hipMalloc((void**)&this->y_vel_after_movement, sizeof(float) * n));

	checkCudaErrors(hipMalloc((void**)&this->types, sizeof(Fishes::FishType) * n));
}

void Fishes::h_CleanMemoryForFishes()
{
	free(this->x_before_movement);
	free(this->y_before_movement);
	free(this->x_vel_before_movement);
	free(this->y_vel_before_movement);

	free(this->x_after_movement);
	free(this->y_after_movement);
	free(this->x_vel_after_movement);
	free(this->y_vel_after_movement);

	free(this->types);
}

void Fishes::d_CleanMemoryForFishes()
{
	checkCudaErrors(hipFree(x_before_movement));
	checkCudaErrors(hipFree(y_before_movement));
	checkCudaErrors(hipFree(x_vel_before_movement));
	checkCudaErrors(hipFree(y_vel_before_movement));

	checkCudaErrors(hipFree(x_after_movement));
	checkCudaErrors(hipFree(y_after_movement));
	checkCudaErrors(hipFree(x_vel_after_movement));
	checkCudaErrors(hipFree(y_vel_after_movement));

	checkCudaErrors(hipFree(types));
}

void Fishes::GenerateRandomFishes(int width, int height, float minVel, float maxVel)
{
	int highWidth = float(width) / 2;
	int lowWidht = -highWidth;
	int highHeight = float(height) / 2;
	int lowHeight = float(height) / 2;
	for (int i = 0; i < this->n; i++)
	{
		this->x_before_movement[i] = rand_float(lowWidht, highWidth);
		this->y_before_movement[i] = rand_float(lowHeight, highHeight);
		// Random normal vector in 2D
		float2 vel = float2();
		vel.x = rand_float(-1.0f, 1.0f);
		vel.y = sqrtf(1.0f - vel.x * vel.x);
		if (rand_float(0.0f, 1.0f) < 0.5f)
		{
			vel.y = -vel.y;
		}
		float velValue = rand_float(minVel, maxVel);
		vel *= velValue;
		this->x_vel_before_movement[i] = vel.x;
		this->y_vel_before_movement[i] = vel.y;
		// TODO: At this moment hardcoded NormalFishes
		this->types[i] = FishType::NormalFish;
	}
}

void Fishes::GenerateTestFishes()
{
	// I'll generate test 20 fishes with same velocity 1 and same vectors of velocity
	for (int i = 0; i < 20; i++)
	{
		this->x_before_movement[i] = -100 + i * 10 - 1;
		this->y_before_movement[i] = -100 + i * 10 - 1;
		this->x_vel_before_movement[i] = 10 * 0.5f;
		this->y_vel_before_movement[i] = 10 * sqrtf(0.75);
		this->types[i] = FishType::NormalFish;
	}
}

void Fishes::d_CopyFishesFromCPU(float* x_before_movement, float* y_before_movement, float* x_vel_before_movement,
	float* y_vel_before_movement, FishType* types)
{
	if (onGpu)
	{
		checkCudaErrors(hipMemcpy(this->x_before_movement, x_before_movement, n * sizeof(float), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(this->y_before_movement, y_before_movement, n * sizeof(float), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(this->x_vel_before_movement, x_vel_before_movement, n * sizeof(float), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(this->y_vel_before_movement, y_vel_before_movement, n * sizeof(float), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(this->types, types, n * sizeof(FishType), hipMemcpyHostToDevice));
	}
}

__host__ __device__ int Fishes::CountForAFish(int index, Grid* grid, Options* options)
{
	float maxVel = options->maxVelNormalFishes;
	float minVel = options->minVelNormalFishes;
	float cohesionNormal = options->cohesionNormalFishes;
	float alignmentNormal = options->alignmentNormalFishes;
	float separationNormal = options->separationNormalFishes;
	int width = options->width;
	int height = options->height;

	int indexOfFish = grid->fish_id[index];
	int indexOfCell = grid->cell_id[index];

	int numberOfCells = grid->ReturnNumberOfCells();
	int numberOfCells_x = grid->ReturnNumberOfCellsX();
	int numberOfCells_y = grid->ReturnNumberOfCellsY();

	int x_ind = indexOfCell % numberOfCells_x;
	int y_ind = indexOfCell / numberOfCells_x;
	// Four cells for each quarter
	int cellsForSearch[4];
	int quarterNumber = grid->quarter_number[index];
	cellsForSearch[0] = indexOfCell;
	int x1;
	int x2;
	int x3;
	int y1;
	int y2;
	int y3;
	int x_rr = (x_ind + 1) % numberOfCells_x;
	int x_ll = (x_ind - 1) >= 0 ? x_ind - 1 : numberOfCells_x - 1;
	int y_tt = (y_ind - 1) >= 0 ? y_ind - 1 : numberOfCells_y - 1;
	int y_bb = (y_ind + 1) % numberOfCells_y;
	// Finding where should we check fishes for interaction
	switch (quarterNumber)
	{
	case 1:
		x1 = x_rr;
		y1 = y_ind;
		x2 = x_rr;
		y2 = y_tt;
		x3 = x_ind;
		y3 = y_tt;
		break;
	case 2:
		x1 = x_ind;
		y1 = y_tt;
		x2 = x_ll;
		y2 = y_tt;
		x3 = x_ll;
		y3 = y_ind;
		break;
	case 3:
		x1 = x_ll;
		y1 = y_ind;
		x2 = x_ll;
		y2 = y_bb;
		x3 = x_ind;
		y3 = y_bb;
		break;
	case 4:
		x1 = x_ind;
		y1 = y_bb;
		x2 = x_rr;
		y2 = y_bb;
		x3 = x_rr;
		y3 = y_ind;
		break;
	default:
		break;
	}
	// Interaction counting 

	float2 velBeforeInteraction = float2();
	velBeforeInteraction.x = x_vel_before_movement[indexOfFish];
	velBeforeInteraction.y = y_vel_before_movement[indexOfFish];
	float2 additionalVel = float2();
	additionalVel.x = 0.0f;
	additionalVel.y = 0.0f;
	// End of interaction counting
	// 
	// finding of direction vector
	float2 velAfterCount = velBeforeInteraction + additionalVel;
	float valueOfVel = cuda_examples::length(velAfterCount);
	float2 directionVect = cuda_examples::normalize(velAfterCount);
	if (valueOfVel > maxVel)
	{
		velAfterCount = directionVect * maxVel;
	}
	else if (valueOfVel < minVel)
	{
		velAfterCount = directionVect * minVel;
	}
	// Adding velocity to position and also adding changing velocity in an array
	float xAfterMovement = x_before_movement[indexOfFish] + velAfterCount.x;
	float yAfterMovement = y_before_movement[indexOfFish] + velAfterCount.y;
	int widthHalf = width / 2;
	int heightHalf = height / 2;
	xAfterMovement = xAfterMovement > widthHalf ? -width + xAfterMovement : xAfterMovement;
	xAfterMovement = xAfterMovement < -widthHalf ? width + xAfterMovement : xAfterMovement;

	yAfterMovement = yAfterMovement > heightHalf ? -height + yAfterMovement : yAfterMovement;
	yAfterMovement = yAfterMovement < -heightHalf ? height + yAfterMovement : yAfterMovement;

	x_after_movement[indexOfFish] = xAfterMovement;
	y_after_movement[indexOfFish] = yAfterMovement;
	x_vel_after_movement[indexOfFish] = velAfterCount.x;
	y_vel_after_movement[indexOfFish] = velAfterCount.y;

	return indexOfFish;
}

__host__ __device__ void Fishes::FindTrianglesForAFish(int index, float* buffer, int lenOfTriang, int widthOfTriang)
{
	float2 vel = float2();
	vel.x = x_after_movement[index];
	vel.y = y_after_movement[index];
	float2 direction = cuda_examples::normalize(vel);
	float2 reversedDirection = -direction;
	float2 normal = float2();
	normal.x = -direction.y;
	normal.y = direction.x;
	float2 first = 3.0f * lenOfTriang * direction / 5.0f;
	float2 second = 2.0f * lenOfTriang * reversedDirection / 5.0f + widthOfTriang * normal / 2.0f;
	float2 third = 2.0f * cuda_examples::dot(direction, second) * direction - second;
	int indexInBuffer = index * 6; // W have 6 elements for each fish
	buffer[indexInBuffer] = first.x;
	buffer[indexInBuffer + 1] = first.y;
	buffer[indexInBuffer + 2] = second.x;
	buffer[indexInBuffer + 3] = second.y;
	buffer[indexInBuffer + 4] = third.x;
	buffer[indexInBuffer + 5] = third.y;
}






